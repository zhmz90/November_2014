#include <stdio.h>
#include <hip/hip_runtime.h>

__global__
void add( int a, int b, int *c ){
	int i = threadIdx.x;
	*(c+i) = a + b;

}

int main(){
	int *h_c;
	int *d_c;
	const int LEN = 100;
	hipMalloc((void **)&d_c, LEN*sizeof(int));
	
	hipEvent_t start, stop;
	hipEventCreate( &start);
	hipEventCreate( &stop);
	hipEventRecord( start, 0);	

	add<<<1,LEN>>>(2, 7, d_c);

	hipEventRecord( stop, 0);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,start,stop);
	hipMemcpy(h_c, d_c, LEN*sizeof(int),hipMemcpyDeviceToHost);
	
//	printf("2 + 7 = %d\n", h_c);
	printf("elapsedTime is: %f", elapsedTime);
	hipFree(d_c);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}

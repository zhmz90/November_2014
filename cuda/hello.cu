#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ 
void hello(int* d_A){
	int i = threadIdx.x;
//	int j = threadIdx.y;
//	d_A[i] = i;
	d_A[i] = i;
	printf("D%d\t",d_A[i]);

}

int main(void){
	
	int *h_A;
	int const LEN = 12;
	size_t const BYTES = LEN*sizeof(int); 
	h_A = (int *)malloc(BYTES);
	int *d_A;
	hipMalloc(&d_A, BYTES);
	hipMemcpy(d_A, h_A, BYTES,hipMemcpyHostToDevice);
	hello<<<1,LEN>>>(d_A);
	hipMemcpy(h_A, d_A, BYTES,hipMemcpyDeviceToHost);
	for (int i=0; i<LEN; i++){
		printf("H %d\t", *h_A);
		h_A++;
	}

//	printf("Hello, world!\n");
	hipFree(d_A);	
	return 0;
}
